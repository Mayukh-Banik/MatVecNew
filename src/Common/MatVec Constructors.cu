#include "MatVecCore/MatVecClassDeclaration.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>

#include "Constants/Common Defs.h"

template <typename T>
MatVec<T>::MatVec(const pybind11::array& array)
{
    if(!array.dtype().is(pybind11::dtype::of<T>()))
    {
        throw std::runtime_error("Input array type was not validated.");
    }
    pybind11::buffer_info buf = array.request();
    this->ndim = static_cast<std::uint64_t>(buf.ndim);
    this->elementCount = 1;
    this->shape.reserve(ndim);
    this->strides.reserve(ndim);
    const std::uint64_t itemsize = sizeof(double);
    if (buf.ndim == 0)
    {
        this->elementCount = 1;
    }
    else if (buf.ndim == 1)
    {
        this->shape = {static_cast<std::uint64_t>(buf.shape[0])};
        this->strides = {itemsize};
        this->elementCount = buf.shape[0];
    }
    else
    {
        for (pybind11::ssize_t i = 0; i < buf.ndim; i++) 
        {
            this->shape.push_back(static_cast<std::uint64_t>(buf.shape[i]));
            this->strides.push_back(static_cast<std::uint64_t>(buf.strides[i]));
            this->elementCount *= this->shape[i];
        }
    }
    this->memSize = this->elementCount * itemsize;
    hipError_t t = hipMalloc((void**) &this->data, this->memSize);
    CUDA_CHECK_ERROR(t);
    t = hipMemcpy(this->data, buf.ptr, this->memSize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(t);
}

template <typename T>
MatVec<T>::MatVec(T* data, std::uint64_t ndim, std::uint64_t elementCount, std::uint64_t memSize, const std::vector<std::uint64_t> shape, const std::vector<std::uint64_t> strides)
{
	this->ndim = ndim;
	this->elementCount = elementCount;
	this->memSize = memSize;
	this->shape = std::vector<std::uint64_t>(shape);
	this->strides = std::vector<std::uint64_t>(strides);
	hipError_t t = hipMemcpy((void**) &this->data, data, this->memSize, hipMemcpyDeviceToDevice);
	CUDA_CHECK_ERROR(t);
}

template <typename T>
MatVec<T>::~MatVec()
{
	hipFree(this->data);
}

// Explicit instantiations for double
template class MatVec<double>;

// template MatVec<double>::MatVec(const pybind11::array& array);
// template MatVec<double>::MatVec(double* data, std::uint64_t ndim, std::uint64_t elementCount, std::uint64_t memSize, const std::vector<std::uint64_t> shape, const std::vector<std::uint64_t> strides);
// template MatVec<double>::~MatVec();


